#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>


__host__
__device__
void vector_mul(float *out, float *a,
        float *b, size_t n) {
    for(size_t i = 0; i < n; i ++){
        out[i] = a[i] * b[i];
    }
}

__global__
void vector_mul_gpu(float* d_out, float* d_a,
        float* d_b, size_t n) {
    vector_mul(d_out, d_a, d_b, n);
}

int main(int argc, char** argv){

    size_t N = 1000'000'000;
    if (argc > 1) {
        N = strtoul(argv[1], 0, 0);
    }

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    std::mt19937 eng{std::random_device()()};
    std::generate(a, a+N, [&eng]() {
        return std::uniform_real_distribution<float>{0.f, 1.f}(eng);
    });

    std::generate(b, b+N, [&eng]() {
        return std::uniform_real_distribution<float>{1.f, 2.f}(eng);
    });


    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing in kernel
    int blockCount = 5;
    int threadCount = 500;
    vector_mul_gpu<<<blockCount, threadCount>>>(d_out, d_a, d_b, N);

    hipDeviceSynchronize();

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // profiling for CPU
    auto s = std::chrono::high_resolution_clock::now();

    vector_mul(out, a, b, N);

    auto d = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - s).count();
    std::cout << d << " ms CPU" << std::endl;

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
