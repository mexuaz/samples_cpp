#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/version.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>

// nvcc --expt-extended-lambda -std=c++14 vector_mul_2.cu -o vector_mul2

int main(int argc, char** argv){

    size_t N = 100'000'000;
    if (argc > 1) {
        N = strtoul(argv[1], 0, 0);
    }

    thrust::host_vector<float> a(N), b(N), out(N);
    thrust::device_vector<float> d_a, d_b, d_out(N);

    // Initialize host arrays
    std::mt19937 eng{std::random_device()()};
    std::generate(a.begin(), a.end(), [&eng]() {
        return std::uniform_real_distribution<float>{0.f, 1.f}(eng);
    });

    std::generate(b.begin(), b.end(), [&eng]() {
        return std::uniform_real_distribution<float>{1.f, 2.f}(eng);
    });

    // Transfer data from host to device memory
    d_a = a;
    d_b = b;
    auto fm = [=]  __device__ (float a, float b) {return a*b;};

    thrust::transform(thrust::cuda::par,
            d_a.begin(), d_a.end(),
            d_b.begin(),
            d_out.begin(), fm);


    // Transfer data back to host memory
    thrust::copy(d_out.cbegin(), d_out.cend(), out.begin());

    // profiling for CPU
    auto s = std::chrono::high_resolution_clock::now();

    std::transform(a.begin(), a.end(),
                      d_b.begin(),
                      d_out.begin(), std::multiplies<float>());

    auto d = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - s).count();
    std::cout << d << " ms CPU" << std::endl;

    return 0;
}
